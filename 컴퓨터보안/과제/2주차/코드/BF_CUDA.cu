#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cstring>
#include <random>
#include <vector>
#include <string>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""

#define PW_LENGTH 8 // 암호 길이
#define NUM_CHARS 94 // 사용 가능한 문자 개수

using namespace std;

vector<string> password_set;
vector<string> character_set = { "0123456789",
                                "!\"#$%&'()*+,-./:;<=.?@[\\]^_`{|}~",
                                "aAbBcCdDeEfFgGhHiIjJkKlLmMnNoOpPqQrRsStTuUvVwWxXyYzZ",
                                "0123456789!\"#$%&'()*+,-./:;<=.?@[\\]^_`{|}~",
                                "0123456789aAbBcCdDeEfFgGhHiIjJkKlLmMnNoOpPqQrRsStTuUvVwWxXyYzZ",
                                "aAbBcCdDeEfFgGhHiIjJkKlLmMnNoOpPqQrRsStTuUvVwWxXyYzZ!\"#$%&'()*+,-./:;<=.?@[\\]^_`{|}~",
                                "0123456789aAbBcCdDeEfFgGhHiIjJkKlLmMnNoOpPqQrRsStTuUvVwWxXyYzZ!\"#$%&'()*+,-./:;<=.?@[\\]^_`{|}~"
};

int Password_character(int size)
{
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<int> digit(0, size - 1);
    int character = digit(gen);
    return character;
}

void Make_password()
{
    string password = "";
    for (int count = 0; count < 10; count++)
    {
        for (int length = 4; length <= 8; length++)
        {
            for (int charset = 0; charset < character_set.size(); charset++)
            {
                for (int index = 0; index < length; index++)
                {
                    password += character_set[charset][Password_character(character_set[charset].length())];
                }
                password_set.push_back(password);
                password.clear();
            }
        }
    }
    random_shuffle(password_set.begin(), password_set.end());
}

__device__ int strncmp(const char* s1, const char* s2, size_t n) {
    unsigned char uc1, uc2;
    if (n == 0)
        return 0;
    while (n-- > 0 && *s1 == *s2) {
        if (n == 0 || *s1 == '\0')
            return 0;
        s1++;
        s2++;
    }
    uc1 = (*(unsigned char*)s1);
    uc2 = (*(unsigned char*)s2);
    return ((uc1 < uc2) ? -1 : (uc1 > uc2));
}
__device__ void my_strcpy(char* s1, char* s2)
{
    int i = 0;
    for (i = 0; s2[i] != '\0'; i++)
        s1[i] = s2[i];
    s1[i] = '\0';
}

__global__ void bruteForceAttackKernel(char* target, char* result, int *length) {
    // 스레드 인덱스 계산
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // 가능한 모든 암호를 생성하여 확인
    char pw[PW_LENGTH + 1];
    pw[PW_LENGTH] = '\0';
    for (int i = 0; i < NUM_CHARS; ++i) {
        for (int j = 0; j < NUM_CHARS; ++j) {
            for (int k = 0; k < NUM_CHARS; ++k) {
                for (int l = 0; l < NUM_CHARS; ++l) {
                    pw[0] = i + 33;
                    pw[1] = j + 33;
                    pw[2] = k + 33;
                    pw[3] = l + 33;
                    if (*length == 4 && strncmp(pw, target, 4) == 0) {
                        my_strcpy(result, pw);
                        return;
                    }
                    if (*length > 4)
                    {
                        for (int m = 0; m < NUM_CHARS; ++m)
                        {
                            pw[4] = m + 33;
                            if (*length == 5 && strncmp(pw, target, 5) == 0) {
                                my_strcpy(result, pw);
                                printf("%s\n", pw);
                                return;
                            }
                            if (*length > 5)
                            {
                                for (int n = 0; n < NUM_CHARS; ++n)
                                {
                                    pw[5] = n + 33;
                                    if (*length == 6 && strncmp(pw, target, 6) == 0) {
                                        my_strcpy(result, pw);
                                        return;
                                    }
                                    if (*length > 6)
                                    {
                                        for (int o = 0; o < NUM_CHARS; ++o)
                                        {
                                            pw[6] = o + 33;
                                            if (*length == 7 && strncmp(pw, target, 7) == 0) {
                                                my_strcpy(result, pw);
                                                return;
                                            }
                                            if (*length > 7)
                                            {
                                                for (int p = 0; p < NUM_CHARS; ++p)
                                                {
                                                    pw[7] = p + 33;
                                                    if (*length == 8 && strncmp(pw, target, 8) == 0) {
                                                        my_strcpy(result, pw);
                                                        return;
                                                    }
                                                }
                                            }
                                        }
                                    }
                                }
                            }
                        }                    
                    }
                }
            }
        }
    }
}

int main() {

    Make_password();
    int count = 0;
    for (auto a : password_set)
    {
        auto start = chrono::high_resolution_clock::now(); //공격 시작 시간 저장
        cout << "Cracking Password " << ++count << ": " << a << endl;   //현재 크래킹할 패스워드 출력
        const char* password = a.c_str();   //패스워드를 const char*에 저장
        int strlen = a.length();  //패스워드 길이  저장
        char target[PW_LENGTH + 1]; // 찾을 암호
        strcpy(target, password);   //패스워드 복사
        char result[PW_LENGTH + 1] = ""; // 찾은 암호
        char* d_target, * d_result; //GPU 메모리에 패스워드 데이터를 옮기기 위한 변수
        int* length;    //GPU 메모리에 패스워드 길이 값을 옮기기 위한 변수
        hipMalloc((void**)&d_target, sizeof(char) * (PW_LENGTH + 1));  //GPU 메모리에 target password를 저장할 메모리 할당
        hipMalloc((void**)&d_result, sizeof(char) * (PW_LENGTH + 1));  //GPU 메모리에 크래킹 후 결과 값을 저장할 메모리 할당
        hipMalloc((void**)&length, sizeof(int));   //GPU 메모리에 패스워드 길이를 저장하기 위한 메모리 할당
        hipMemcpy(d_target, target, sizeof(char) * (PW_LENGTH + 1), hipMemcpyHostToDevice);   //target password를 GPU 메모리에 복사
        hipMemcpy(length, &strlen, sizeof(int), hipMemcpyHostToDevice);   //패스워드 길이를 GPU 메모리에 복사

        int block_size = 256;   //block 사이즈 설정
        int grid_size = (NUM_CHARS * NUM_CHARS * NUM_CHARS * NUM_CHARS + block_size - 1) / block_size; //grid 사이즈 설정
        bruteForceAttackKernel <<<grid_size, block_size >> > (d_target, d_result, length);  //GPU에서 Brute Force Attack 실행
        hipMemcpy(result, d_result, sizeof(char) * (PW_LENGTH + 1), hipMemcpyDeviceToHost); //결과 값을 다시 CPU 메모리에 복사
        hipFree(d_target); //GPU 메모리 반환
        hipFree(d_result); //GPU 메모리 반환
        std::cout << "Found password: " << result << endl; //결과 값 출력
        auto stop = chrono::high_resolution_clock::now(); //공격 마친 시간 저장
        auto duration = chrono::duration_cast<chrono::seconds>(stop - start); //소요 시간 계산
        cout << "Duration of time : " << duration.count() << " seconds" << endl; //소요시간 출력
        cout << "#############################################" << endl << endl;
    }
    return 0;
}