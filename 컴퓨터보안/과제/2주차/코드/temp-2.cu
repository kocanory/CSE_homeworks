#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <cstring>
#include <random>
#include <vector>
#include <string>
#include <chrono>
#include "hip/hip_runtime.h"
#include ""

#define PW_LENGTH 8 // 암호 길이
#define NUM_CHARS 94 // 사용 가능한 문자 개수

using namespace std;

vector<string> password_set;
vector<string> character_set = { "0123456789",
                                "!\"#$%&'()*+,-./:;<=.?@[\\]^_`{|}~",
                                "aAbBcCdDeEfFgGhHiIjJkKlLmMnNoOpPqQrRsStTuUvVwWxXyYzZ",
                                "0123456789!\"#$%&'()*+,-./:;<=.?@[\\]^_`{|}~",
                                "0123456789aAbBcCdDeEfFgGhHiIjJkKlLmMnNoOpPqQrRsStTuUvVwWxXyYzZ",
                                "aAbBcCdDeEfFgGhHiIjJkKlLmMnNoOpPqQrRsStTuUvVwWxXyYzZ!\"#$%&'()*+,-./:;<=.?@[\\]^_`{|}~",
                                "0123456789aAbBcCdDeEfFgGhHiIjJkKlLmMnNoOpPqQrRsStTuUvVwWxXyYzZ!\"#$%&'()*+,-./:;<=.?@[\\]^_`{|}~"
};

int Password_character(int size)
{
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<int> digit(0, size - 1);
    int character = digit(gen);
    return character;
}

void Make_password()
{
    string password = "";
    for (int count = 0; count < 10; count++)
    {
        for (int length = 4; length <= 8; length++)
        {
            for (int charset = 0; charset < character_set.size(); charset++)
            {
                for (int index = 0; index < length; index++)
                {
                    password += character_set[charset][Password_character(character_set[charset].length())];
                }
                password_set.push_back(password);
                password.clear();
            }
        }
    }
    //random_device rd;
    //mt19937 gen(rd());
    //shuffle(password_set.begin(), password_set.end(), gen);
}

__device__ int strncmp(const char* s1, const char* s2, size_t n) {
    unsigned char uc1, uc2;
    /* Nothing to compare?  Return zero.  */
    if (n == 0)
        return 0;
    /* Loop, comparing bytes.  */
    while (n-- > 0 && *s1 == *s2) {
        /* If we've run out of bytes or hit a null, return zero
        since we already know *s1 == *s2.  */
        if (n == 0 || *s1 == '\0')
            return 0;
        s1++;
        s2++;
    }
    uc1 = (*(unsigned char*)s1);
    uc2 = (*(unsigned char*)s2);
    return ((uc1 < uc2) ? -1 : (uc1 > uc2));
}
__device__ void my_strcpy(char* s1, char* s2)
{
    int i = 0;
    for (i = 0; s2[i] != '\0'; i++)
        s1[i] = s2[i];
    s1[i] = '\0';
}

__global__ void recur(char * target, char* result, int *length, int *count)
{
    if (*length > *count - 1)
    {
        for (int m = 0; m < NUM_CHARS; ++m)
        {
            pw[&count - 1] = m + 33;
            if (*length == *count && strncmp(pw, target, *count) == 0) {
                my_strcpy(result, pw);
                printf("%s\n", pw);
                return;
            }
            *count += 1;
            if(*length > count) recur(target, result, length, count);
            *count -= 1;
        }
    }
}
// CUDA 커널 함수
__global__ void bruteForceAttackKernel(char* target, char* result, int *length, int *count) {
    // 스레드 인덱스 계산
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // 가능한 모든 암호를 생성하여 확인
    char pw[PW_LENGTH + 1];
    pw[PW_LENGTH] = '\0';
    for (int i = 0; i < NUM_CHARS; ++i) {
        for (int j = 0; j < NUM_CHARS; ++j) {
            for (int k = 0; k < NUM_CHARS; ++k) {
                for (int l = 0; l < NUM_CHARS; ++l) {
                    pw[0] = i + 33;
                    pw[1] = j + 33;
                    pw[2] = k + 33;
                    pw[3] = l + 33;
                    if (*length == 4 && strncmp(pw, target, 4) == 0) {
                        my_strcpy(result, pw);
                        return;
                    }
                    recur(target, result, length, count);
                }
            }
        }
    }
}

int main() {

    Make_password();
    int count = 0;
    for (auto a : password_set)
    {
        auto start = chrono::high_resolution_clock::now();
        cout << "Cracking Password " << ++count << ": " << a << endl;
        const char* password = a.c_str();
        int strlen = a.length(), cnt = 5;
        char target[PW_LENGTH + 1]; // 찾을 암호
        strcpy(target, password);
        char result[PW_LENGTH + 1] = ""; // 찾은 암호
        char* d_target, * d_result;
        int* length, *count;
        hipMalloc((void**)&d_target, sizeof(char) * (PW_LENGTH + 1));
        hipMalloc((void**)&d_result, sizeof(char) * (PW_LENGTH + 1));
        hipMalloc((void**)&length, sizeof(int));
        hipMalloc((void**)&count, sizeof(int));
        hipMemcpy(d_target, target, sizeof(char) * (PW_LENGTH + 1), hipMemcpyHostToDevice);
        hipMemcpy(length, &strlen, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(count, &cnt, sizeof(int), hipMemcpyHostToDevice);

        int block_size = 256;
        int grid_size = (NUM_CHARS * NUM_CHARS * NUM_CHARS * NUM_CHARS + block_size - 1) / block_size;
        bruteForceAttackKernel <<<grid_size, block_size >> > (d_target, d_result, length, count);
        hipMemcpy(result, d_result, sizeof(char) * (PW_LENGTH + 1), hipMemcpyDeviceToHost);
        hipFree(d_target);
        hipFree(d_result);
        std::cout << "Found password: " << result << endl;
        auto stop = chrono::high_resolution_clock::now();
        auto duration = chrono::duration_cast<chrono::seconds>(stop - start);
        cout << "Duration of time : " << duration.count() << " seconds" << endl;
        cout << "#############################################" << endl << endl;
    }
    return 0;
}